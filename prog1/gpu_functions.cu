
#include <hip/hip_runtime.h>

/**
 * @brief Applies the Gauss Elimination Formula
 * 
 * @param kj 
 * @param ki_ii 
 * @param ij 
 */
__device__ void formulaGPU (double *kj, double ki_ii, double ij)
{
    *kj = *kj - (ki_ii * ij);
}


/**
 * @brief Swap columns on gpu
 * 
 * @param mat 
 * @param x 
 * @param y 
 */
__device__ void swapColsOnGPU(double *mat, int x, int y){
    
    double aux = mat[x];
    mat[x] = mat[y];
    mat[y] = aux;
}


/**
 * @brief Calculate the determinats of matrices on GPU
 * 
 * @param mat 
 * @param determinant_results 
 * @param sector_size 
 */
__global__ static void determinantOnGPU (double * __restrict__ mat, double * __restrict__ determinant_results,  int sector_size)
{   
 
    int bx = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
    int idx = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;

    mat += bx * sector_size * sector_size;

    
    for (int j = 0; j<=idx; j++) {

        /* check if diagonal is 0 to change column*/
        if (mat[j*sector_size + j] == 0) {
            int colSwap = -1;

            for (int i=j+1; i<sector_size; i++) {
                if (mat[j*sector_size + i] != 0.0) {
                colSwap = i;
                }
            }

            if (colSwap == -1) {
                determinant_results[bx] = 0;
                break;
            } else {
                swapColsOnGPU(mat, j + idx * sector_size, colSwap + idx * sector_size);
            }

            __syncthreads(); 
        }

        // Apply formula process
        for (int i = j+1; i<sector_size; i++) {
            
            /*get the division value before apply on formula*/
            double division =  mat[i*sector_size + j] / mat[j*sector_size + j];

            __syncthreads(); 

            // Apply formula
            formulaGPU(&mat[i*sector_size + idx], division, mat[j*sector_size+idx]);
        }
        
        if (idx == j) {
            determinant_results[bx] = determinant_results[bx] * mat[ (idx*sector_size) + idx ];
        }
    }

}

